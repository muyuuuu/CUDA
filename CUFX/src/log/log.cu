#include "hip/hip_runtime.h"
#include "log.cuh"

void ErrorBackTrace(hipError_t status_code, const char *file, int line_idx) {
    if (status_code != hipSuccess) {
        printf("CUDA ERROR: \n \t code = %d\n\t name = %s\n\t desc = %s\n\t file = %s\n\t line = %d\n", status_code,
               hipGetErrorName(status_code), hipGetErrorString(status_code), file, line_idx);
    }
}
