#include "testcase.cuh"

#define TEST_TABLE X_MACROS(TEST_REDUCT_SUM, TestReductSum)

/*定义命令列表*/
typedef enum {
    TEST_START,
#define X_MACROS(a, b) a,
    TEST_TABLE
#undef X_MACROS
        TEST_END
} cmd_e;

/*定义字符串列表用作Log打印*/
const char *cmd_str[] = {
#define X_MACROS(a, b) #a,
    TEST_TABLE
#undef X_MACROS
};

typedef int (*func)(void);

const func func_table[] = {
#define X_MACROS(a, b) b,
    TEST_TABLE
#undef X_MACROS
};

static int cmd_handle(int cmd) {
    if (cmd < TEST_END) {
        return func_table[cmd]();
    }
    return -1;
}

int main() {
    printf("=============== summary report ===================\n");
    int total = TEST_END - 1;
    int passed = 0;

    for (int i = TEST_START; i < TEST_END; i++) {
        if ((i > TEST_START) && (i < TEST_END)) {
            if (0 == cmd_handle(i - 1)) {
                passed++;
                printf("=============== Case: %d TEST Passed ===================\n", i);
            } else {
                printf("=============== Case: %d TEST Falied ===================\n", i);
            }
        }
    }
    printf("=============== total: %d ===================\n", total);
    printf("=============== pass : %d ===================\n", passed);
    printf("=============== fail: %d ===================\n", total - passed);

    return 0;
}