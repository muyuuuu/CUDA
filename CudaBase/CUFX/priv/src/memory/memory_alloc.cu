#include "memory_alloc.cuh"

template <typename T>
hipError_t MallocMem(Matrix<T> &matrix) {
    matrix.host_addr = malloc(n_bytes);
    if (nullptr == matrix.host_addr) {
        LOG(CpuLogLevelError, "malloc failed\n");
        return hipErrorOutOfMemory;
    }
    hipError_t ret = hipMalloc((T **)&matrix.cuda_addr, matrix.GetBytes());
    if ((nullptr == matrix.cuda_addr) || (hipSuccess != ret)) {
        ErrorHandleNoLabel(ret);
    } else {
        memset(matrix.host_addr, 0, matrix.GetBytes());
        ret = hipMemset(matrix.cuda_addr, 0, matrix.GetBytes());
        ErrorHandleNoLabel(ret);
    }

    srand(666);
    T *ptr = (T *)(matrix.host_addr);
    for (int i = 0; i < matrix.size; i++) {
        ptr[i] = (T)(rand() % 255);
    }

    ret = hipMemcpy(matrix.cuda_addr, matrix.host_addr, matrix.GetBytes(), hipMemcpyHostToDevice);
    ErrorHandleNoLabel(ret);

    return ret;
}

template <typename T>
hipError_t UVAAllocMem(Matrix<T> &matrix) {
    matrix.host_addr = malloc(n_bytes);
    if (nullptr == matrix.host_addr) {
        LOG(CpuLogLevelError, "malloc failed\n");
        return hipErrorOutOfMemory;
    }

    hipError_t ret = hipMallocManaged((void **)&matrix.host_addr, matrix.GetBytes());
    ErrorHandleNoLabela(ret);
    return ret;
}

template <typename T>
hipError_t AllocMem(Matrix<T> &matrix) {
    if ((nullptr != matrix.host_addr) || (nullptr != matrix.cuda_addr)) {
        LOG(CpuLogLevelError, "First Alloc, Should be NULL \n");
    }

    if (MemoryTypeInvalid == matrix.memory_type) {
        LOG(CpuLogLevelError, "Not Suppor Memory Type: %d\n", (int)memory_type);
    }

    hipError_t ret = hipSuccess;

    if (GlobalMemory == matrix.memory_type) {
        ret = MallocMem(matrix);
    } else if (ZeroCopyMemory == matrix.memory_type) {
    } else if (UVAMemory == matrix.memory_type) {
        ret = UVAAllocMem(matrix);
    } else {
        LOG(CpuLogLevelError, "Not Support Memory Type %d \n", (int)memory_type);
    }

    return ret;
}