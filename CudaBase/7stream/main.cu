#include "hip/hip_runtime.h"
#include <cstdio>

#include "../tools/common.cuh"
#include "../tools/matrix.cuh"

const int global_size = 512000;
const int local_size = 1024;
const int num_tasks = 10;

template <typename T, size_t size, int N>
__global__ void MatrixAddScalar(T *src) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) { return; }

    src[idx] += N * tan((float)(N / 10));
}

__global__ void kernel_1() {
    double sum = 0.0;
    for (int i = 0; i < global_size; i++) sum = sum + tan(0.1) * tan(0.1);
}

__global__ void kernel_2() {
    double sum = 0.0;
    for (int i = 0; i < global_size; i++) sum = sum + tan(0.1) * tan(0.1);
}

__global__ void kernel_3() {
    double sum = 0.0;
    for (int i = 0; i < global_size; i++) sum = sum + tan(0.1) * tan(0.1);
}

__global__ void kernel_4() {
    double sum = 0.0;
    for (int i = 0; i < global_size; i++) sum = sum + tan(0.1) * tan(0.1);
}

void Func1() {
    hipError_t ret;
    TMatrix<int> src;
    const int step = global_size / num_tasks;

    const size_t n_bytes = sizeof(int) * global_size;
    src.host_addr = (int *)malloc(n_bytes);
    ret = hipMalloc((int **)&src.cuda_addr, n_bytes);
    ErrorHandleNoLabel(ret);

    hipStream_t stream_arr[num_tasks];

    for (int i = 0; i < num_tasks; i++) { hipStreamCreate(&(stream_arr[i])); }

    int *data = (int *)src.host_addr;
    for (int i = 0; i < global_size; i++) { data[i] = rand() % 255; }

    printf(" src[277] = %d\n", data[277]);

    hipEvent_t start, stop;
    ret = hipEventCreate(&start);
    ErrorHandleNoLabel(ret);
    ret = hipEventCreate(&stop);
    ret = hipEventRecord(start);
    ErrorHandleNoLabel(ret);

    for (int i = 0; i < num_tasks; i++) {
        int offset = i * step;

        ret = hipMemcpyAsync(src.cuda_addr + offset, src.host_addr + offset, step, hipMemcpyHostToDevice,
                              stream_arr[i]);
        ErrorHandleNoLabel(ret);

        MatrixAddScalar<int, step, 13><<<step, local_size, 0, stream_arr[i]>>>(src.cuda_addr + offset);

        ret = hipMemcpyAsync(src.host_addr + offset, src.cuda_addr + offset, step, hipMemcpyDeviceToHost,
                              stream_arr[i]);
        ErrorHandleNoLabel(ret);
    }

    for (int i = 0; i < num_tasks; i++) { ret = hipStreamSynchronize(stream_arr[i]); }

    hipEventRecord(stop);
    // CPU Tasks
    ret = hipEventSynchronize(stop);
    float elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(" Block MatrixAddScalar elapsed time: %.4f ms\n", elapsed_time);

    for (int i = 0; i < num_tasks; i++) {
        ret = hipStreamDestroy(stream_arr[i]);
        ErrorHandleNoLabel(ret);
    }
    printf(" src[277] = %d\n", data[277]);

    ret = hipEventRecord(start);
    MatrixAddScalar<int, global_size, 13><<<global_size, local_size>>>(src.cuda_addr);
    ret = hipEventRecord(stop);
    ret = hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(" Total MatrixAddScalar elapsed time: %.4f ms\n", elapsed_time);

    free(src.host_addr);
    src.host_addr = nullptr;

    hipFree(src.cuda_addr);
    src.cuda_addr = nullptr;
}

void Func2() {
    hipError_t ret;
    hipStream_t stream_arr[num_tasks];
    for (int i = 0; i < num_tasks; i++) { hipStreamCreate(&(stream_arr[i])); }

    hipEvent_t wait_tokens[num_tasks];
    for (int i = 0; i < num_tasks; i++) { hipEventCreateWithFlags(&wait_tokens[i], hipEventDisableTiming); }

    hipEvent_t start, stop;
    ret = hipEventCreate(&start);
    ErrorHandleNoLabel(ret);
    ret = hipEventCreate(&stop);
    ret = hipEventRecord(start);

    for (int i = 0; i < num_tasks; i++) {
        kernel_1<<<1, 1, 0, stream_arr[i]>>>();
        kernel_2<<<1, 1, 0, stream_arr[i]>>>();
        kernel_3<<<1, 1, 0, stream_arr[i]>>>();

        hipEventRecord(wait_tokens[i]);
        hipStreamWaitEvent(stream_arr[i], wait_tokens[i], 0);

        kernel_4<<<1, 1, 0, stream_arr[i]>>>();
    }
    hipEventRecord(stop);
    // CPU Tasks
    ret = hipEventSynchronize(stop);
    ErrorHandleNoLabel(ret);

    for (int i = 0; i < num_tasks; i++) {
        ret = hipStreamDestroy(stream_arr[i]);
        ErrorHandleNoLabel(ret);
    }

    float elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(" Func2 elapsed time: %.4f ms\n", elapsed_time);
}

// 回调函数原型，必须符合 hipStreamCallback_t 的定义
void CUDART_CB my_callback(hipStream_t stream, hipError_t status, void *userData) {
    int *p = (int *)userData;
    if (status == hipSuccess) {
        printf("Callback from stream %d, Wait Event Success \n", *p);
    } else {
        printf("Error in stream callback: %s\n", hipGetErrorString(status));
    }
}

void Func3() {
    hipError_t ret;
    hipStream_t stream_arr[num_tasks];
    for (int i = 0; i < num_tasks; i++) { hipStreamCreate(&(stream_arr[i])); }

    hipEvent_t wait_tokens[num_tasks];
    for (int i = 0; i < num_tasks; i++) { hipEventCreateWithFlags(&wait_tokens[i], hipEventDisableTiming); }

    hipEvent_t start, stop;
    ret = hipEventCreate(&start);
    ErrorHandleNoLabel(ret);
    ret = hipEventCreate(&stop);
    ret = hipEventRecord(start);

    int stream_idx[num_tasks];

    for (int i = 0; i < num_tasks; i++) {
        stream_idx[i] = i;
        kernel_1<<<1, 1, 0, stream_arr[i]>>>();
        kernel_2<<<1, 1, 0, stream_arr[i]>>>();
        kernel_3<<<1, 1, 0, stream_arr[i]>>>();

        hipEventRecord(wait_tokens[i]);
        hipStreamWaitEvent(stream_arr[i], wait_tokens[i], 0);

        kernel_4<<<1, 1, 0, stream_arr[i]>>>();
        hipStreamAddCallback(stream_arr[i], my_callback, stream_idx + i, 0);
    }
    hipEventRecord(stop);
    // CPU Tasks ...
    ret = hipEventSynchronize(stop);
    ErrorHandleNoLabel(ret);

    float elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(" Func3 elapsed time: %.4f ms\n", elapsed_time);
}

int main() {
    Func1();
    Func2();
    Func3();
    return 0;
}